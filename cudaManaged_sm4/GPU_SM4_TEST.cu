#include "GPU_SM4.h"

#define test_blocks (1<<25)

int main()
{
	u1 * key;
	hipMallocManaged(&key,sizeof(u1)*16);
	u1 * p;
	hipMallocManaged(&p,sizeof(u1)*16*test_blocks);

	u1 *c;
	hipMallocManaged(&c,sizeof(u1)*16*test_blocks);

	u1 p2[SM4_BLOCK_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};

	u1 key2[SM4_KEY_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};
	for (int i=0; i<16; i++)
	{
		key[i] = key2[i];
	}

  for (int i=0; i<test_blocks*16; i++)
  {
    p[i] = p2[i%16];
  }

  gpu_sm4_encrypt(p,key,c,4);

  outputChar(c, 16*4);

  gpu_sm4_decrypt(p,key,c,4);

  outputChar(p, 16*4);

	//benchmark
	benchmark_sm4_encrypt(p,key,c,1<<4);
	benchmark_sm4_decrypt(p,key,c,1<<4);

	benchmark_sm4_encrypt(p,key,c,1<<8);
	benchmark_sm4_decrypt(p,key,c,1<<8);

	benchmark_sm4_encrypt(p,key,c,1<<10);
	benchmark_sm4_decrypt(p,key,c,1<<10);

	benchmark_sm4_encrypt(p,key,c,1<<12);
	benchmark_sm4_decrypt(p,key,c,1<<12);

	benchmark_sm4_encrypt(p,key,c,1<<16);
	benchmark_sm4_decrypt(p,key,c,1<<16);

	benchmark_sm4_encrypt(p,key,c,1<<18);
	benchmark_sm4_decrypt(p,key,c,1<<18);

	benchmark_sm4_encrypt(p,key,c,1<<20);
	benchmark_sm4_decrypt(p,key,c,1<<20);

	benchmark_sm4_encrypt(p,key,c, 1 << 23);
	benchmark_sm4_decrypt(p,key,c, 1 << 23);
}
