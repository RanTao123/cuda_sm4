#include "hip/hip_runtime.h"
#include "GPU_SM4.h"

#define test_blocks (1<<25)

void __global__ add(const int* x, const int* y, int* z, const int N)
{
	const int n = blockDim.x * blockIdx.x + threadIdx.x;

	if (n < N)
	{
		for (int i = 0; i < 10000; ++i)
		{
			z[n] = x[n] + y[n];
		}
	}
}
int main()
{
	
	//u1 * key;
	//hipMallocManaged(&key,sizeof(u1)*16);
	u1* key = new u1[sizeof(u1) * 16];
	//u1 * p;
	//hipMallocManaged(&p,sizeof(u1)*16*test_blocks);
	u1* p = new u1[sizeof(u1) * 16 * test_blocks];
	//u1 *c;
	//hipMallocManaged(&c,sizeof(u1)*16*test_blocks);
	u1* c = new u1[sizeof(u1) * 16 * test_blocks];


	u1 p2[SM4_BLOCK_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};

	u1 key2[SM4_KEY_SIZE] = {0x01, 0x23, 0x45, 0x67, 0x89, 0xab, 0xcd, 0xef, 0xfe, 0xdc, 0xba, 0x98, 0x76, 0x54, 0x32, 0x10};
	for (int i=0; i<16; i++)
	{
		key[i] = key2[i];
	}

  for (int i=0; i<test_blocks*16; i++)
  {
    p[i] = p2[i%16];
  }
  u1* d_key;
  u1* d_p;
  u1* d_c;
  hipMalloc((void**)&d_key, sizeof(u1) * 16);
  hipMalloc((void**)&d_p, sizeof(u1) * 16 * test_blocks);
  hipMalloc((void**)&d_c, sizeof(u1) * 16 * test_blocks);
  hipMemcpy(d_key, key, sizeof(u1) * 16, hipMemcpyHostToDevice);
  hipMemcpy(d_p, p, sizeof(u1) * 16 * test_blocks, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, sizeof(u1) * 16 * test_blocks, hipMemcpyHostToDevice);
  gpu_sm4_encrypt(d_p,d_key,d_c,4);
  hipMemcpy(c, d_c, sizeof(u1) * 16 * test_blocks, hipMemcpyDeviceToHost);
  outputChar(c, 16*4);

  gpu_sm4_decrypt(d_p,d_key,d_c,4);
  hipMemcpy(p, d_p, sizeof(u1) * 16 * test_blocks, hipMemcpyDeviceToHost);
  outputChar(p, 16*4);

	//benchmark

	int* d_x, * d_y, * d_z;
	
	
  
  
	
	//double tt = (double)(elapsed_time) / (100);
	//double speed = (double)(16 * 1 << 4) / (1024 * 1024 * tt);
	//printf("SM4_encrypt>>> blocks: %d, time: %f s, speed: %f MB/s\n", 1 << 4, tt, speed);
	benchmark_sm4_encrypt(p, key, c, d_p, d_key, d_c, 1 << 4);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c,1<<4);

	benchmark_sm4_encrypt(p,key,c, d_p, d_key, d_c,1<<8);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c,1<<8);

	benchmark_sm4_encrypt(p,key,c, d_p, d_key, d_c,1<<10);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c,1<<10);

	benchmark_sm4_encrypt(p,key,c, d_p, d_key, d_c,1<<12);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c,1<<12);

	benchmark_sm4_encrypt(p,key,c, d_p, d_key, d_c,1<<16);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c,1<<16);

	benchmark_sm4_encrypt(p,key,c, d_p, d_key, d_c,1<<18);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c,1<<18);

	benchmark_sm4_encrypt(p,key,c, d_p, d_key, d_c,1<<20);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c,1<<20);

	benchmark_sm4_encrypt(p,key,c, d_p, d_key, d_c, 1 << 25);
	benchmark_sm4_decrypt(p,key,c, d_p, d_key, d_c, 1 << 25);
}
